#include "hip/hip_runtime.h"
#include<common.h>
#include<timer.h>

#define BLOCK_DIM 1024

// kogge stone
__global__ void scan_kernel(float* input,float* output,float* partialSums,unsigned int N){

    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    __shared__ float buffer_s[BLOCK_DIM];

    buffer_s[threadIdx.X] = input[I];

    for(const int stride = 1;stride <= BLOCK_DIM / 2;stride *= 2){
        if(threadIdx.x >= stride){
            float temp;
            temp =  buffer_s[threadIdx.x] + buffer_s[threadIdx - stride];
            __syncthreads();
            buffer_s[threadIdx.x] = temp;
        }
           
        __syncthreads();
    }

    if(threadIdx.x == BLOCK_DIM - 1)
        partialSums[blockIdx.x] = buffer_s[threadIdx.x];

    output[i] = buffer_s[threadIdx.x];
}

// 利用双缓冲解决读写冲突假依赖
__global__ void scan_kernel_db(float* input,float* output,float* partialSums,unsigned int N){

    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    __shared__ float buffer1_s[BLOCK_DIM];
    __shared__ float buffer2_s[BLOCK_DIM];

    float* inBuffer_s = buffer1_s;
    float* outBuffer_s = buffer2_s;

    inBuffer_s[threadIdx.X] = input[I];
    __syncthreads();

    for(const int stride = 1;stride <= BLOCK_DIM / 2;stride *= 2){
        if(threadIdx.x >= stride){
            outBuffer_s[threadIdx.x] =  inBuffer_s[threadIdx.x] + inBuffer_s[threadIdx - stride];
        }else   
            outBuffer_s[threadIdx.x] = inBuffer_s[threadIdx.x];
           
        __syncthreads();
        float* temp = inBuffer_s;
        inBuffer_s = outBuffer_s;
        outBuffer_s = temp;
    }

    if(threadIdx.x == BLOCK_DIM - 1)
        partialSums[blockIdx.x] = inBuffer_s[threadIdx.x];

    output[i] = inBuffer_s[threadIdx.x];
}

// brent kung reduce control divergence
__global__ void scan_kernel_bk(Float* input, Float* output, Float* partialSums, unsigned int N) {
    unsigned int segment = blockIdx.x * blockDim.x * 2;
    __shared__ float buffer_s[2 * BLOCK_DIM];

    buffer_s[threadIdx.x] = input[segment + threadIdx.x];
    buffer_s[threadIdx.x + BLOCK_DIM] = input[segment + threadIdx.x + BLOCK_DIM];
    __syncthreads();

    // Reduction phase
    for (unsigned int stride = 1; stride <= BLOCK_DIM; stride *= 2) {
        unsigned int i = (threadIdx.x + 1) * 2 * stride - 1;
        if (i < 2 * BLOCK_DIM) {
            buffer_s[i] += buffer_s[i - stride];
        }
        __syncthreads();
    }

    // Post reduction
    for (unsigned int stride = BLOCK_DIM / 2; stride >= 1; stride /= 2) {
        unsigned int i = (threadIdx.x + 1) * 2 * stride - 1;
        if (i + stride < 2 * BLOCK_DIM) {
            buffer_s[i + stride] += buffer_s[i];
        }
        __syncthreads();
    }

    if (threadIdx.x == BLOCK_DIM - 1) {
        partialSums[blockIdx.x] = buffer_s[2 * BLOCK_DIM - 1];
    }

    output[segment + threadIdx.x] = buffer_s[threadIdx.x];
    output[segment + threadIdx.x + BLOCK_DIM] = buffer_s[threadIdx.x + BLOCK_DIM];
}

// segmented scan
__global__ void scan_kernel(float* input, float* output, float* partialSums, unsigned int N) {
    unsigned int segment = blockIdx.x * blockDim.x * COARSE_FACTOR;

    // Load elements from global memory to shared memory
    __shared__ float buffer_s[BLOCK_DIM * COARSE_FACTOR];
    for(unsigned int c = 0; c < COARSE_FACTOR; ++c) {
        buffer_s[c * BLOCK_DIM + threadIdx.x] = input[segment + c * BLOCK_DIM + threadIdx.x];
    }
    __syncthreads();

    // Thread scan
    unsigned int threadSegment = threadIdx.x * COARSE_FACTOR;
    for(unsigned int c = 1; c < COARSE_FACTOR; ++c) {
        buffer_s[threadSegment + c] += buffer_s[threadSegment + c - 1];
    }
    __syncthreads();

    // Allocate and initialize double buffers for partial sums
    __shared__ float buffer1_s[BLOCK_DIM];
    __shared__ float buffer2_s[BLOCK_DIM];
    float* inBuffer_s = buffer1_s;
    float* outBuffer_s = buffer2_s;

    inBuffer_s[threadIdx.x] = buffer_s[threadSegment + COARSE_FACTOR - 1];
    __syncthreads();

    // Parallel scan of partial sums
    for(unsigned int stride = 1; stride <= BLOCK_DIM / 2; stride *= 2) {
        if(threadIdx.x >= stride) {
            outBuffer_s[threadIdx.x] = inBuffer_s[threadIdx.x] + inBuffer_s[threadIdx.x - stride];
        } else {
            outBuffer_s[threadIdx.x] = inBuffer_s[threadIdx.x];
        }
        __syncthreads();
        float* tmp = inBuffer_s;
        inBuffer_s = outBuffer_s;
        outBuffer_s = tmp;
    }

    // Add previous thread's partial sum
    if (threadIdx.x > 0) {
        float prevPartialSum = inBuffer_s[threadIdx.x - 1];
        for (unsigned int c = 0; c < COARSE_FACTOR; ++c) {
            buffer_s[threadSegment + c] += prevPartialSum;
        }
    }
    __syncthreads();

    // Save block's partial sum
    if (threadIdx.x == BLOCK_DIM - 1) {
        partialSums[blockIdx.x] = inBuffer_s[threadIdx.x];
    }

    // Write output
    for (unsigned int c = 0; c < COARSE_FACTOR; ++c) {
        output[segment + c * BLOCK_DIM + threadIdx.x] = buffer_s[c * BLOCK_DIM + threadIdx.x];
    }

}

// single kernel scan
